#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <algorithm>
#include <chrono>
#include <cinttypes>
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <thread>
#include <numeric>
#include <string>
#include <functional>
#include <map>

#include "benchmark_util.cuh"    
#include "merlin_hashtable.cuh"  
#include "ycsb.cuh"

using namespace nv::merlin;
using benchmark::Timer;



// Custom string type that is compatible with CUDA and Merlin library
struct CustomString {
  char data[MAX_STRING_LENGTH];
  
  __host__ __device__ CustomString() {
    memset(data, 0, MAX_STRING_LENGTH);
  }
  
  __host__ __device__ CustomString(const CustomString& other) {
    memcpy(data, other.data, MAX_STRING_LENGTH);
  }
  
  __host__ __device__ CustomString& operator=(const CustomString& other) {
    if (this != &other) {
      memcpy(data, other.data, MAX_STRING_LENGTH);
    }
    return *this;
  }
  
  __host__ __device__ CustomString& operator=(const char* str) {
    if (str) {
      size_t len = 0;
      while (str[len] && len < MAX_STRING_LENGTH - 1) len++;
      memcpy(data, str, len);
      data[len] = '\0';
    } else {
      data[0] = '\0';
    }
    return *this;
  }
  
  __host__ __device__ CustomString& operator+=(const CustomString& other) {
    size_t len = 0;
    while (data[len] && len < MAX_STRING_LENGTH - 1) len++;
    
    size_t i = 0;
    while (other.data[i] && len + i < MAX_STRING_LENGTH - 1) {
      data[len + i] = other.data[i];
      i++;
    }
    data[len + i] = '\0';
    return *this;
  }
  
  __host__ __device__ char& operator[](size_t idx) {
    return data[idx];
  }
  
  __host__ __device__ const char& operator[](size_t idx) const {
    return data[idx];
  }
};

static void usage(const char* prog, const bool is_error = false) {
  std::string usage_msg = "Usage: " + std::string(prog) + " [options]\n"
                         + "Options:\n"
                         + "  -h, --help             print this help message\n"
                         + "  --gpu_id=N             GPU ID (default 0)\n"
                         + "  --recordcount=N        preload N records (default 6M)\n"
                         + "  --num_batch_ops=N      number of batch operations to execute (default 10)\n"
                         + "  --readproportion=f     read ratio 0‑1 (default 1.0)\n"
                         + "  --distribution=d       zipf | uniform (default zipf)\n"
                         + "  --theta=f              zipf theta (default 0.0)\n"
                         + "  --multiget_batch=N     keys per read batch (default 64K)\n"
                         + "  --multiset_batch=N     keys per write batch (default 64K)\n"
                         + "  --dim=N                vector dimension (default 5)\n"
                         + "  --initcapacity=N       table capacity (default 12M)\n"
                         + "  --hbm_gb=N             HBM budget in GB (default 18)\n"
                         + "  --seed=N               RNG seed (default 42)\n"
                         + "  --num_streams=N        Number of CUDA streams (default 1)\n";
  if (is_error) {
    std::cerr << "Error: " << prog << " [options]\n"
              << usage_msg << std::endl;
  } else {
    std::cout << usage_msg << std::endl;
  }
}

Flags parse_flags(int argc, char** argv) {
  Flags f;
  static struct option long_opts[] = {
      {"help",            no_argument,       nullptr, 'h'  },
      {"gpu_id",          required_argument, nullptr, 'i'  },
      {"recordcount",     required_argument, nullptr, 'r'  },
      {"num_batch_ops",   required_argument, nullptr, 'o'  },
      {"readproportion",  required_argument, nullptr, 'p'  },
      {"distribution",    required_argument, nullptr, 'd'  },
      {"theta",           required_argument, nullptr, 't'  },
      {"multiget_batch",  required_argument, nullptr, 'g'  },
      {"multiset_batch",  required_argument, nullptr, 's'  },
      {"dim",             required_argument, nullptr, 'm'  },
      {"initcapacity",    required_argument, nullptr, 'c'  },
      {"hbm_gb",          required_argument, nullptr, 'b'  },
      {"seed",            required_argument, nullptr, 'e'  },
      {"num_streams",     required_argument, nullptr, 'N'  },
      {0,0,0,0}
  };
  int opt;
  int long_idx;
  while ((opt = getopt_long(argc, argv, "h", long_opts, &long_idx)) != -1) {
    switch (opt) {
      case 'h': usage(argv[0], false); exit(EXIT_SUCCESS);
      case 'i': f.gpu_id          = std::strtoul (optarg, nullptr, 10); break;
      case 'r': f.record_count    = std::strtoull(optarg, nullptr, 10);   break;
      case 'o': f.num_batch_ops   = std::strtoull(optarg, nullptr, 10);   break;
      case 'p': f.read_prop       = std::strtod (optarg, nullptr);        break;
      case 'd': f.distribution    = optarg;                               break;
      case 't': f.theta           = std::strtod (optarg, nullptr);        break;
      case 'g': f.multiget_batch_size = std::strtoul (optarg, nullptr, 10);   break;
      case 's': f.multiset_batch_size = std::strtoul (optarg, nullptr, 10);   break;
      case 'm': f.dim             = std::strtoul (optarg, nullptr, 10);   break;
      case 'c': f.init_capacity   = std::strtoull(optarg, nullptr, 10);   break;
      case 'b': f.hbm_gb          = std::strtoul (optarg, nullptr, 10);   break;
      case 'e': f.seed            = std::strtoull(optarg, nullptr, 10);   break;
      case 'N': f.num_streams     = std::strtoul (optarg, nullptr, 10);   break;
      default : usage(argv[0], true); exit(EXIT_FAILURE);
    }
  }
  if (f.read_prop < 0.0 || f.read_prop > 1.0) {
    std::cerr << "readproportion must be in [0,1]\n"; exit(EXIT_FAILURE);
  }
  return f;
}

// -------------------------------------------------------------
// Key generators
// -------------------------------------------------------------
class KeyGenerator {
 public:
  virtual uint64_t operator()() = 0;
  virtual std::unique_ptr<KeyGenerator> clone() const = 0; 
  virtual ~KeyGenerator() = default;
};

class UniformKeyGen : public KeyGenerator {
  std::uniform_int_distribution<uint64_t> dist_;
  std::mt19937_64 eng_;
 public:
  UniformKeyGen(uint64_t max_key, uint64_t seed): dist_(0, max_key-1), eng_(seed) {}
  uint64_t operator()() override { return dist_(eng_); }

  std::unique_ptr<KeyGenerator> clone() const override {
    return std::make_unique<UniformKeyGen>(*this);
  }
};

// Approximate Zipf generator using rejection sampling.
class ZipfKeyGen : public KeyGenerator {
  uint64_t n_;              // number of items
  uint64_t base_;           // min value
  double theta_;            // zipfian constant
  double alpha_;            // computed from theta
  double zeta_n_;          // zeta(n)
  double zeta2_theta_;     // zeta(2,theta)
  double eta_;             // computed from theta
  std::mt19937_64 eng_;    // random number generator
  std::uniform_real_distribution<double> uni_;

  double zeta(uint64_t n, double theta, uint64_t start = 0, double initialSum = 0) {
    double sum = initialSum;
    for(uint64_t i = start; i < n; i++) {
      sum += 1.0 / std::pow(i + 1, theta);
    }
    return sum;
  }

 public:
  ZipfKeyGen(uint64_t n, double theta, uint64_t seed)
    : n_(n), base_(0), theta_(theta), eng_(seed), uni_(0.0, 1.0) {
    // Compute zeta values
    zeta_n_ = zeta(n_, theta_);
    zeta2_theta_ = zeta(2, theta_);
    alpha_ = 1.0 / (1.0 - theta_);
    eta_ = (1.0 - std::pow(2.0/n_, 1.0 - theta_)) / (1.0 - zeta2_theta_ / zeta_n_);
  }

  uint64_t operator()() override {
    double u = uni_(eng_);
    double uz = u * zeta_n_;

    if (uz < 1.0) return base_;
    if (uz < 1.0 + std::pow(0.5, theta_)) return base_ + 1;

    uint64_t ret = base_ + static_cast<uint64_t>(
      n_ * std::pow(eta_ * u - eta_ + 1, alpha_)
    );
    return std::min(ret, n_ - 1);  // ensure we don't exceed range
  }

  std::unique_ptr<KeyGenerator> clone() const override {
    return std::make_unique<ZipfKeyGen>(*this);
  }
};

using K = uint64_t;
using S = uint64_t;
using V = CustomString;
using HKVTable = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;
using TableOptions = nv::merlin::HashTableOptions;


struct DeviceBuffers {
  K* d_keys;
  V* d_values;
  V* d_values_out;
  S* d_scores;
  bool* d_found;
  V** d_values_ptr;

  explicit DeviceBuffers(uint32_t batch_size, uint32_t dim) {
    CUDA_CHECK(hipMalloc(&d_keys,        batch_size * sizeof(K)));
    CUDA_CHECK(hipMalloc(&d_values,      batch_size * dim * sizeof(V)));
    CUDA_CHECK(hipMalloc(&d_values_out,  batch_size * dim * sizeof(V)));
    CUDA_CHECK(hipMalloc(&d_scores,      batch_size * sizeof(S)));
    CUDA_CHECK(hipMalloc(&d_found,       batch_size * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_values_ptr,  batch_size * sizeof(V*)));
  }
  ~DeviceBuffers() {
    hipFree(d_keys);
    hipFree(d_values);
    hipFree(d_values_out);
    hipFree(d_scores);
    hipFree(d_found);
    hipFree(d_values_ptr);
  }
};


void generate_keys_parallel(uint32_t                 bs,
                            const KeyGenerator&      keygen_proto,
                            std::vector<K>&          out_keys,
                            unsigned                 T = std::thread::hardware_concurrency())
{
    // print the number of threads
    // std::cout << "T: " << T << std::endl;
    std::vector<std::vector<K>> local_keys(T);   

    const uint32_t chunk = (bs + T - 1) / T;     
    auto worker = [&](unsigned tid) {
        uint32_t begin = tid * chunk;
        uint32_t end   = std::min(begin + chunk, bs);
        if (begin >= end) return;               

        auto kg = keygen_proto.clone();        

        local_keys[tid].reserve(end - begin);
        for (uint32_t i = begin; i < end; ++i) {
            local_keys[tid].push_back((*kg)()); 
        }
    };

    // Start threads
    std::vector<std::thread> pool;
    for (unsigned t = 0; t < T; ++t)
        pool.emplace_back(worker, t);
    for (auto& th : pool)
        th.join();                 

    out_keys.clear();
    size_t total = 0;
    for (const auto& v : local_keys) total += v.size();
    out_keys.reserve(total);

    for (auto& v : local_keys) {
        out_keys.insert(out_keys.end(),
                        std::make_move_iterator(v.begin()),
                        std::make_move_iterator(v.end()));
    }
}


BenchmarkResult run_ycsb(const Flags& cfg) {
  // Set the GPU device first
  CUDA_CHECK(hipSetDevice(cfg.gpu_id));

  int device_count = 0;
  hipGetDeviceCount(&device_count);
  std::cout << "Available CUDA devices: " << device_count << std::endl;

  std::cout << "GPU: " << cfg.gpu_id << std::endl;
  // ----- Create key generator -----
  std::unique_ptr<KeyGenerator> keygen;
  if (cfg.distribution == "uniform") {
    keygen = std::make_unique<UniformKeyGen>(cfg.record_count, cfg.seed);
  } else if (cfg.distribution == "zipf") {
    keygen = std::make_unique<ZipfKeyGen>(cfg.record_count, cfg.theta, cfg.seed);
  } else {
    std::cerr << "Unsupported distribution: " << cfg.distribution << std::endl;
    return {0.0, 0.0, 0.0};
  }

  // ----- Initialize table -----
  TableOptions options;
  options.init_capacity = cfg.init_capacity;
  options.max_capacity  = cfg.init_capacity;
  options.dim           = cfg.dim;
  options.max_hbm_for_vectors = nv::merlin::GB(cfg.hbm_gb);
  options.io_by_cpu     = false;
  options.device_id     = cfg.gpu_id;  // Set the device ID in options

  std::unique_ptr<HKVTable> table = std::make_unique<HKVTable>();
  table->init(options);

  // ----- Allocate host buffers -----
  std::vector<K> h_keys(cfg.multiset_batch_size); // Use multiset_batch_size for write operations
  std::vector<CustomString> h_vals(cfg.multiset_batch_size * cfg.dim);
  std::mt19937 rng(cfg.seed); 
  
  // Host storage for preloaded data for verification
  std::map<K, std::vector<CustomString>> preloaded_data_host;

  // Characters for random string generation
  const std::string charset = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
  std::uniform_int_distribution<size_t> char_dist(0, charset.size() - 1);
  
  // ----- Device buffers -----
  // Ensure dbuf is large enough for both read and write batches in the workload phase
  uint32_t max_op_batch_size = std::max(cfg.multiget_batch_size, cfg.multiset_batch_size);

  std::vector<DeviceBuffers> stream_specific_dbufs;
  for (uint32_t i = 0; i < cfg.num_streams; ++i) {
    stream_specific_dbufs.emplace_back(max_op_batch_size, cfg.dim);
  }
  std::cout << "Created " << cfg.num_streams << " stream-specific device buffers." << std::endl;

  // Create a vector of CUDA streams
  std::vector<hipStream_t> streams(cfg.num_streams);
  for (uint32_t i = 0; i < cfg.num_streams; ++i) {
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }
  hipStream_t default_stream_for_preload = streams[0]; 

  // ------------------------------------------------------
  // Preload phase (multiset / insert_or_assign)
  // ------------------------------------------------------
  std::cout << "Preloading " << cfg.record_count << " records..." << std::endl;
  uint64_t inserted = 0;
  while (inserted < cfg.record_count) {
    uint32_t this_batch = std::min<uint64_t>(cfg.multiset_batch_size, cfg.record_count - inserted);
    // Generate keys
    for(uint32_t i=0; i<this_batch; ++i) h_keys[i] = inserted + i;
    
    // Generate values
    for(uint32_t i=0; i<this_batch*cfg.dim; ++i) {
      CustomString& str = h_vals[i];
      memset(str.data, 0, MAX_STRING_LENGTH);
      
      for (uint32_t j = 0; j < MAX_STRING_LENGTH-1; ++j) {
        str[j] = charset[char_dist(rng)];
      }

      str[MAX_STRING_LENGTH-1] = '\0';
    }
    
    for (uint32_t k_idx = 0; k_idx < this_batch; ++k_idx) {
        K current_key = h_keys[k_idx];
        std::vector<CustomString> vals_for_key(cfg.dim);
        for (uint32_t d = 0; d < cfg.dim; ++d) {
            vals_for_key[d] = h_vals[k_idx * cfg.dim + d];
        }
        preloaded_data_host[current_key] = vals_for_key;
    }

    CUDA_CHECK(hipMemcpy(stream_specific_dbufs[0].d_keys, h_keys.data(), this_batch*sizeof(K), hipMemcpyHostToDevice)); 
    CUDA_CHECK(hipMemcpy(stream_specific_dbufs[0].d_values, h_vals.data(), this_batch*cfg.dim*sizeof(V), hipMemcpyHostToDevice));
    table->insert_or_assign(this_batch, stream_specific_dbufs[0].d_keys, stream_specific_dbufs[0].d_values, nullptr, default_stream_for_preload);
    inserted += this_batch;
    if (inserted % (1024ull * 1024ull) == 0) {
      std::cout << "Preloaded " << inserted << " records..." << std::endl;
    }
  }

  CUDA_CHECK(hipStreamSynchronize(default_stream_for_preload));

  // ------------------------------------------------------
  // Workload phase
  // ------------------------------------------------------
  std::cout << "Executing " << cfg.num_batch_ops << " batch operations..." << std::endl;
  double total_time = 0;
  Timer<double> timer;
  uint64_t total_ops = 0;

  // Create a vector to determine the type of each batch (true for read, false for write)
  std::vector<bool> batch_types(cfg.num_batch_ops);
  uint64_t num_read_batches = static_cast<uint64_t>(cfg.num_batch_ops * cfg.read_prop);
  std::fill(batch_types.begin(), batch_types.begin() + num_read_batches, true);
  std::fill(batch_types.begin() + num_read_batches, batch_types.end(), false);
  
  // Shuffle the batch types
  std::mt19937_64 rng_batch(cfg.seed + 1); 
  std::shuffle(batch_types.begin(), batch_types.end(), rng_batch);

  // --- Pre-generate all keys, values, and workload operations ---
  std::cout << "Pre-generating all keys, values, and operations for " << cfg.num_batch_ops << " batches..." << std::endl;
  std::vector<std::vector<K>> all_batch_keys(cfg.num_batch_ops);
  std::vector<std::vector<CustomString>> all_batch_values(cfg.num_batch_ops); 
  std::vector<std::function<void()>> workload_batch_fns;
  workload_batch_fns.reserve(cfg.num_batch_ops);

  // GPU storage for all read results
  std::vector<V*> d_all_results_values_out_list; 
  std::vector<bool*> d_all_results_found_list;    
  std::vector<uint32_t> read_op_batch_indices;    
  std::vector<uint32_t> read_op_actual_sizes;   

  uint32_t current_stream_idx = 0; // Initialize stream index for round-robin assignment

  for (uint64_t batch_idx = 0; batch_idx < cfg.num_batch_ops; ++batch_idx) {
    bool is_read_op = batch_types[batch_idx];
    uint32_t current_batch_size = is_read_op ? cfg.multiget_batch_size : cfg.multiset_batch_size;
    total_ops += current_batch_size; // Calculate total_ops here, outside the timed section
    

    generate_keys_parallel(current_batch_size, *keygen, all_batch_keys[batch_idx]);

    if (!is_read_op) { // It's a write batch, generate values
        all_batch_values[batch_idx].resize(current_batch_size * cfg.dim);
        for (uint32_t i = 0; i < current_batch_size * cfg.dim; ++i) {
            CustomString& str = all_batch_values[batch_idx][i];
            for (uint32_t j = 0; j < MAX_STRING_LENGTH - 1; ++j) {
                str[j] = charset[char_dist(rng)];
            }
            str[MAX_STRING_LENGTH - 1] = '\0';
        }
    }

    // Create and store the lambda for the current batch operation
    hipStream_t captured_stream = streams[current_stream_idx];
    DeviceBuffers& captured_dbuf = stream_specific_dbufs[current_stream_idx];

    if (is_read_op) {
        // Allocate GPU memory for this specific read batch's results
        V* d_current_batch_values_out;
        bool* d_current_batch_found;
        CUDA_CHECK(hipMalloc(&d_current_batch_values_out, current_batch_size * cfg.dim * sizeof(V)));
        CUDA_CHECK(hipMalloc(&d_current_batch_found, current_batch_size * sizeof(bool)));
        
        d_all_results_values_out_list.push_back(d_current_batch_values_out);
        d_all_results_found_list.push_back(d_current_batch_found);
        read_op_batch_indices.push_back(batch_idx);
        read_op_actual_sizes.push_back(current_batch_size);

        workload_batch_fns.emplace_back(
            [&keys = all_batch_keys[batch_idx], 
             table_ptr = table.get(),
             stream = captured_stream,
             &dbuf_ref = captured_dbuf, // Stream-specific working buffer
             d_target_values_out = d_current_batch_values_out, // Destination for this batch's values
             d_target_found = d_current_batch_found]           // Destination for this batch's found flags
            () { 
                CUDA_CHECK(hipMemcpyAsync(dbuf_ref.d_keys, keys.data(), keys.size()*sizeof(K), hipMemcpyHostToDevice, stream));
                table_ptr->find(keys.size(), dbuf_ref.d_keys, d_target_values_out, d_target_found, nullptr, stream);
            }
        );
    } else { // Write op
        workload_batch_fns.emplace_back(
            [&keys = all_batch_keys[batch_idx], 
             &vals = all_batch_values[batch_idx], 
             dim_val = cfg.dim, 
             table_ptr = table.get(),
             stream = captured_stream,
             &dbuf_ref = captured_dbuf] // Stream-specific working buffer
            () { 
                CUDA_CHECK(hipMemcpyAsync(dbuf_ref.d_keys, keys.data(), keys.size()*sizeof(K), hipMemcpyHostToDevice, stream));
                CUDA_CHECK(hipMemcpyAsync(dbuf_ref.d_values, vals.data(), keys.size()*dim_val*sizeof(V), hipMemcpyHostToDevice, stream));
                table_ptr->insert_or_assign(keys.size(), dbuf_ref.d_keys, dbuf_ref.d_values, nullptr, stream);
            }
        );
    }
    current_stream_idx = (current_stream_idx + 1) % cfg.num_streams;
  }
  std::cout << "Pre-generation complete." << std::endl;
  
  // --- Start Timer for Workload ---
  timer.start();
  

  for (uint64_t batch_op_idx = 0; batch_op_idx < cfg.num_batch_ops; ++batch_op_idx) {
    workload_batch_fns[batch_op_idx]();
  }
  
  // --- Synchronize and Stop Timer ---
  for (uint32_t i = 0; i < cfg.num_streams; ++i) {
    CUDA_CHECK(hipStreamSynchronize(streams[i]));
  }
  timer.end();
  total_time = timer.getResult();
  double secs = total_time;
  double ops_per_sec = total_ops / secs;
  double gbkv_per_sec = total_ops / secs / 1e9;

  std::cout << "total_ops,time_ms,ops_per_sec,GB-kv/s\n";
  std::cout << total_ops << "," << secs*1000.0 << "," << ops_per_sec << "," << gbkv_per_sec << std::endl;

  // --- Verification Phase ---
  // std::cout << "Copying results from GPU to Host for verification..." << std::endl;
  // std::vector<std::vector<CustomString>> host_all_read_values_out(d_all_results_values_out_list.size());
  // std::vector<std::vector<unsigned char>> host_all_read_found_uchar(d_all_results_found_list.size());

  // for (size_t i = 0; i < d_all_results_values_out_list.size(); ++i) {
  //     uint32_t num_keys_in_batch = read_op_actual_sizes[i];

  //     host_all_read_values_out[i].resize(num_keys_in_batch * cfg.dim);
  //     host_all_read_found_uchar[i].resize(num_keys_in_batch);

  //     CUDA_CHECK(hipMemcpy(host_all_read_values_out[i].data(),
  //                           d_all_results_values_out_list[i],
  //                           num_keys_in_batch * cfg.dim * sizeof(V),
  //                           hipMemcpyDeviceToHost));
  //     CUDA_CHECK(hipMemcpy(host_all_read_found_uchar[i].data(),
  //                           d_all_results_found_list[i],
  //                           num_keys_in_batch * sizeof(bool),
  //                           hipMemcpyDeviceToHost));
  // }
  // std::cout << "Copying complete. Starting verification..." << std::endl;
  
  // uint64_t correct_reads_found = 0;
  // uint64_t correct_reads_not_found = 0;
  // uint64_t incorrect_reads_value_mismatch = 0;
  // uint64_t incorrect_reads_should_be_found = 0;
  // uint64_t incorrect_reads_should_not_be_found = 0;
  // uint64_t total_read_keys_verified = 0;

  // for (size_t i = 0; i < host_all_read_values_out.size(); ++i) { // Loop through read batches
  //     uint32_t original_batch_idx = read_op_batch_indices[i];
  //     const std::vector<K>& current_read_op_keys = all_batch_keys[original_batch_idx];
  //     const std::vector<CustomString>& actual_values_out_flat = host_all_read_values_out[i];
  //     const std::vector<unsigned char>& actual_found_flags_uchar = host_all_read_found_uchar[i];

  //     for (size_t k_idx = 0; k_idx < current_read_op_keys.size(); ++k_idx) { // Loop through keys in batch
  //         total_read_keys_verified++;
  //         K key_queried = current_read_op_keys[k_idx];
  //         bool found_in_table = (actual_found_flags_uchar[k_idx] != 0);

  //         auto it_preloaded = preloaded_data_host.find(key_queried);
  //         bool was_preloaded = (it_preloaded != preloaded_data_host.end());

  //         if (found_in_table) {
  //             if (was_preloaded) {
  //                 bool match = true;
  //                 const std::vector<CustomString>& expected_vals_for_key = it_preloaded->second;
  //                 for (uint32_t d = 0; d < cfg.dim; ++d) {
  //                     if (memcmp(actual_values_out_flat[k_idx * cfg.dim + d].data, expected_vals_for_key[d].data, MAX_STRING_LENGTH) != 0) {
  //                         match = false;
  //                         break;
  //                     }
  //                 }
  //                 if (match) {
  //                     correct_reads_found++;
  //                 } else {
  //                     incorrect_reads_value_mismatch++;
  //                     // Optional: Print mismatch details
  //                     // std::cerr << "Mismatch for key " << key_queried << std::endl;
  //                 }
  //             } else {
  //                 incorrect_reads_should_not_be_found++;
  //                 // std::cerr << "Key " << key_queried << " found but was not preloaded." << std::endl;
  //             }
  //         } else { // Not found in table
  //             if (was_preloaded) {
  //                 incorrect_reads_should_be_found++;
  //                 // std::cerr << "Key " << key_queried << " was preloaded but not found." << std::endl;
  //             } else {
  //                 correct_reads_not_found++;
  //             }
  //         }
  //     }
  // }
  // std::cout << "Verification Complete:" << std::endl;
  // std::cout << "  Total read keys verified: " << total_read_keys_verified << std::endl;
  // std::cout << "  Correct reads found: " << correct_reads_found << std::endl;
  // std::cout << "  Correct reads not found: " << correct_reads_not_found << std::endl;
  // std::cout << "  Incorrect - Value Mismatch: " << incorrect_reads_value_mismatch << std::endl;
  // std::cout << "  Incorrect - Should Have Been Found (but wasn't): " << incorrect_reads_should_be_found << std::endl;
  // std::cout << "  Incorrect - Should Not Have Been Found (but was): " << incorrect_reads_should_not_be_found << std::endl;


  // // Clean up GPU memory for results
  // for (V* ptr : d_all_results_values_out_list) {
  //     if (ptr) CUDA_CHECK(hipFree(static_cast<void*>(ptr)));
  // }
  // for (bool* ptr : d_all_results_found_list) {
  //     if (ptr) CUDA_CHECK(hipFree(static_cast<void*>(ptr)));
  // }
  // d_all_results_values_out_list.clear();
  // d_all_results_found_list.clear();

  stream_specific_dbufs.clear();
 
  for (uint32_t i = 0; i < cfg.num_streams; ++i) {

    CUDA_CHECK(hipStreamDestroy(streams[i]));
  }


  return {secs, ops_per_sec, gbkv_per_sec};
}