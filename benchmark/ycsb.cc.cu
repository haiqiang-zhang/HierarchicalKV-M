#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <algorithm>
#include <chrono>
#include <cinttypes>
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <thread>
#include <numeric>
#include <string>

#include "benchmark_util.cuh"    
#include "merlin_hashtable.cuh"  
#include "ycsb.cuh"

using namespace nv::merlin;
using benchmark::Timer;

// Define the fixed string length
const size_t MAX_STRING_LENGTH = 10; // Adjust this size as needed

// Custom string type that is compatible with CUDA and Merlin library
struct CustomString {
  char data[MAX_STRING_LENGTH];
  
  __host__ __device__ CustomString() {
    memset(data, 0, MAX_STRING_LENGTH);
  }
  
  __host__ __device__ CustomString(const CustomString& other) {
    memcpy(data, other.data, MAX_STRING_LENGTH);
  }
  
  __host__ __device__ CustomString& operator=(const CustomString& other) {
    if (this != &other) {
      memcpy(data, other.data, MAX_STRING_LENGTH);
    }
    return *this;
  }
  
  __host__ __device__ CustomString& operator=(const char* str) {
    if (str) {
      size_t len = 0;
      while (str[len] && len < MAX_STRING_LENGTH - 1) len++;
      memcpy(data, str, len);
      data[len] = '\0';
    } else {
      data[0] = '\0';
    }
    return *this;
  }
  
  __host__ __device__ CustomString& operator+=(const CustomString& other) {
    size_t len = 0;
    while (data[len] && len < MAX_STRING_LENGTH - 1) len++;
    
    size_t i = 0;
    while (other.data[i] && len + i < MAX_STRING_LENGTH - 1) {
      data[len + i] = other.data[i];
      i++;
    }
    data[len + i] = '\0';
    return *this;
  }
  
  __host__ __device__ char& operator[](size_t idx) {
    return data[idx];
  }
  
  __host__ __device__ const char& operator[](size_t idx) const {
    return data[idx];
  }
};

static void usage(const char* prog, const bool is_error = false) {
  std::string usage_msg = "Usage: " + std::string(prog) + " [options]\n"
                         + "Options:\n"
                         + "  -h, --help             print this help message\n"
                         + "  --gpu_id=N             GPU ID (default 0)\n"
                         + "  --recordcount=N        preload N records (default 6M)\n"
                         + "  --num_batch_ops=N      number of batch operations to execute (default 10)\n"
                         + "  --readproportion=f     read ratio 0‑1 (default 1.0)\n"
                         + "  --distribution=d       zipf | uniform (default zipf)\n"
                         + "  --theta=f              zipf theta (default 0.0)\n"
                         + "  --multiget_batch=N     keys per read batch (default 64K)\n"
                         + "  --multiset_batch=N     keys per write batch (default 64K)\n"
                         + "  --dim=N                vector dimension (default 5)\n"
                         + "  --initcapacity=N       table capacity (default 12M)\n"
                         + "  --hbm_gb=N             HBM budget in GB (default 18)\n"
                         + "  --seed=N               RNG seed (default 42)\n";
  if (is_error) {
    std::cerr << "Error: " << prog << " [options]\n"
              << usage_msg << std::endl;
  } else {
    std::cout << usage_msg << std::endl;
  }
}

Flags parse_flags(int argc, char** argv) {
  Flags f;
  static struct option long_opts[] = {
      {"help",            no_argument,       nullptr, 'h'  },
      {"gpu_id",          required_argument, nullptr, 'i'  },
      {"recordcount",     required_argument, nullptr, 'r'  },
      {"num_batch_ops",   required_argument, nullptr, 'o'  },
      {"readproportion",  required_argument, nullptr, 'p'  },
      {"distribution",    required_argument, nullptr, 'd'  },
      {"theta",           required_argument, nullptr, 't'  },
      {"multiget_batch",  required_argument, nullptr, 'g'  },
      {"multiset_batch",  required_argument, nullptr, 's'  },
      {"dim",             required_argument, nullptr, 'm'  },
      {"initcapacity",    required_argument, nullptr, 'c'  },
      {"hbm_gb",          required_argument, nullptr, 'b'  },
      {"seed",            required_argument, nullptr, 'e'  },
      {0,0,0,0}
  };
  int opt;
  int long_idx;
  while ((opt = getopt_long(argc, argv, "h", long_opts, &long_idx)) != -1) {
    switch (opt) {
      case 'h': usage(argv[0], false); exit(EXIT_SUCCESS);
      case 'i': f.gpu_id          = std::strtoul (optarg, nullptr, 10); break;
      case 'r': f.record_count    = std::strtoull(optarg, nullptr, 10);   break;
      case 'o': f.num_batch_ops   = std::strtoull(optarg, nullptr, 10);   break;
      case 'p': f.read_prop       = std::strtod (optarg, nullptr);        break;
      case 'd': f.distribution    = optarg;                               break;
      case 't': f.theta           = std::strtod (optarg, nullptr);        break;
      case 'g': f.multiget_batch_size = std::strtoul (optarg, nullptr, 10);   break;
      case 's': f.multiset_batch_size = std::strtoul (optarg, nullptr, 10);   break;
      case 'm': f.dim             = std::strtoul (optarg, nullptr, 10);   break;
      case 'c': f.init_capacity   = std::strtoull(optarg, nullptr, 10);   break;
      case 'b': f.hbm_gb          = std::strtoul (optarg, nullptr, 10);   break;
      case 'e': f.seed            = std::strtoull(optarg, nullptr, 10);   break;
      default : usage(argv[0], true); exit(EXIT_FAILURE);
    }
  }
  if (f.read_prop < 0.0 || f.read_prop > 1.0) {
    std::cerr << "readproportion must be in [0,1]\n"; exit(EXIT_FAILURE);
  }
  return f;
}

// -------------------------------------------------------------
// Key generators
// -------------------------------------------------------------
class KeyGenerator {
 public:
  virtual uint64_t operator()() = 0;
  virtual std::unique_ptr<KeyGenerator> clone() const = 0; 
  virtual ~KeyGenerator() = default;
};

class UniformKeyGen : public KeyGenerator {
  std::uniform_int_distribution<uint64_t> dist_;
  std::mt19937_64 eng_;
 public:
  UniformKeyGen(uint64_t max_key, uint64_t seed): dist_(0, max_key-1), eng_(seed) {}
  uint64_t operator()() override { return dist_(eng_); }

  std::unique_ptr<KeyGenerator> clone() const override {
    return std::make_unique<UniformKeyGen>(*this);
  }
};

// Approximate Zipf generator using rejection sampling.
class ZipfKeyGen : public KeyGenerator {
  uint64_t n_;              // number of items
  uint64_t base_;           // min value
  double theta_;            // zipfian constant
  double alpha_;            // computed from theta
  double zeta_n_;          // zeta(n)
  double zeta2_theta_;     // zeta(2,theta)
  double eta_;             // computed from theta
  std::mt19937_64 eng_;    // random number generator
  std::uniform_real_distribution<double> uni_;

  double zeta(uint64_t n, double theta, uint64_t start = 0, double initialSum = 0) {
    double sum = initialSum;
    for(uint64_t i = start; i < n; i++) {
      sum += 1.0 / std::pow(i + 1, theta);
    }
    return sum;
  }

 public:
  ZipfKeyGen(uint64_t n, double theta, uint64_t seed)
    : n_(n), base_(0), theta_(theta), eng_(seed), uni_(0.0, 1.0) {
    // Compute zeta values
    zeta_n_ = zeta(n_, theta_);
    zeta2_theta_ = zeta(2, theta_);
    alpha_ = 1.0 / (1.0 - theta_);
    eta_ = (1.0 - std::pow(2.0/n_, 1.0 - theta_)) / (1.0 - zeta2_theta_ / zeta_n_);
  }

  uint64_t operator()() override {
    double u = uni_(eng_);
    double uz = u * zeta_n_;

    if (uz < 1.0) return base_;
    if (uz < 1.0 + std::pow(0.5, theta_)) return base_ + 1;

    uint64_t ret = base_ + static_cast<uint64_t>(
      n_ * std::pow(eta_ * u - eta_ + 1, alpha_)
    );
    return std::min(ret, n_ - 1);  // ensure we don't exceed range
  }

  std::unique_ptr<KeyGenerator> clone() const override {
    return std::make_unique<ZipfKeyGen>(*this);
  }
};

using K = uint64_t;
using S = uint64_t;
using V = CustomString;
using HKVTable = nv::merlin::HashTable<K, V, S, EvictStrategy::kLru>;
using TableOptions = nv::merlin::HashTableOptions;


struct DeviceBuffers {
  K* d_keys;
  V* d_values;
  V* d_values_out;
  S* d_scores;
  bool* d_found;
  V** d_values_ptr;

  explicit DeviceBuffers(uint32_t batch_size, uint32_t dim) {
    CUDA_CHECK(hipMalloc(&d_keys,        batch_size * sizeof(K)));
    CUDA_CHECK(hipMalloc(&d_values,      batch_size * dim * sizeof(V)));
    CUDA_CHECK(hipMalloc(&d_values_out,  batch_size * dim * sizeof(V)));
    CUDA_CHECK(hipMalloc(&d_scores,      batch_size * sizeof(S)));
    CUDA_CHECK(hipMalloc(&d_found,       batch_size * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_values_ptr,  batch_size * sizeof(V*)));
  }
  ~DeviceBuffers() {
    hipFree(d_keys);
    hipFree(d_values);
    hipFree(d_values_out);
    hipFree(d_scores);
    hipFree(d_found);
    hipFree(d_values_ptr);
  }
};


void generate_keys_parallel(uint32_t                 bs,
                            const KeyGenerator&      keygen_proto,
                            std::vector<K>&          out_keys,
                            unsigned                 T = std::thread::hardware_concurrency())
{
    // print the number of threads
    std::cout << "T: " << T << std::endl;
    std::vector<std::vector<K>> local_keys(T);   

    const uint32_t chunk = (bs + T - 1) / T;     
    auto worker = [&](unsigned tid) {
        uint32_t begin = tid * chunk;
        uint32_t end   = std::min(begin + chunk, bs);
        if (begin >= end) return;               

        auto kg = keygen_proto.clone();        

        local_keys[tid].reserve(end - begin);
        for (uint32_t i = begin; i < end; ++i) {
            local_keys[tid].push_back((*kg)()); 
        }
    };

    // Start threads
    std::vector<std::thread> pool;
    for (unsigned t = 0; t < T; ++t)
        pool.emplace_back(worker, t);
    for (auto& th : pool)
        th.join();                 

    out_keys.clear();
    size_t total = 0;
    for (const auto& v : local_keys) total += v.size();
    out_keys.reserve(total);

    for (auto& v : local_keys) {
        out_keys.insert(out_keys.end(),
                        std::make_move_iterator(v.begin()),
                        std::make_move_iterator(v.end()));
    }
}


BenchmarkResult run_ycsb(const Flags& cfg) {
  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, cfg.gpu_id));
  std::cout << "GPU: " << props.pciDeviceID << std::endl;
  // ----- Create key generator -----
  std::unique_ptr<KeyGenerator> keygen;
  if (cfg.distribution == "uniform") {
    keygen = std::make_unique<UniformKeyGen>(cfg.record_count, cfg.seed);
  } else if (cfg.distribution == "zipf") {
    keygen = std::make_unique<ZipfKeyGen>(cfg.record_count, cfg.theta, cfg.seed);
  } else {
    std::cerr << "Unsupported distribution: " << cfg.distribution << std::endl;
    return {0.0, 0.0, 0.0};
  }

  // ----- Initialize table -----
  TableOptions options;
  options.init_capacity = cfg.init_capacity;
  options.max_capacity  = cfg.init_capacity;
  options.dim           = cfg.dim;
  options.max_hbm_for_vectors = nv::merlin::GB(cfg.hbm_gb);
  options.io_by_cpu     = false;

  std::unique_ptr<HKVTable> table = std::make_unique<HKVTable>();
  table->init(options);

  // ----- Allocate host buffers -----
  std::vector<K> h_keys(cfg.multiset_batch_size); // Use multiset_batch_size for write operations
  std::vector<CustomString> h_vals(cfg.multiset_batch_size * cfg.dim);
  std::mt19937 rng(cfg.seed);  // Use the same seed for reproducibility
  
  // Characters for random string generation
  const std::string charset = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
  std::uniform_int_distribution<size_t> char_dist(0, charset.size() - 1);
  
  // ----- Device buffers -----
  DeviceBuffers dbuf(cfg.multiset_batch_size, cfg.dim); // Use multiset_batch_size for write operations
  hipStream_t stream; CUDA_CHECK(hipStreamCreate(&stream));

  // ------------------------------------------------------
  // Preload phase (multiset / insert_or_assign)
  // ------------------------------------------------------
  std::cout << "Preloading " << cfg.record_count << " records..." << std::endl;
  uint64_t inserted = 0;
  while (inserted < cfg.record_count) {
    uint32_t this_batch = std::min<uint64_t>(cfg.multiset_batch_size, cfg.record_count - inserted);
    // Generate sequential keys
    for(uint32_t i=0; i<this_batch; ++i) h_keys[i] = inserted + i;
    
    // Generate random string values
    for(uint32_t i=0; i<this_batch*cfg.dim; ++i) {
      CustomString& str = h_vals[i];
      // Clear the string buffer
      memset(str.data, 0, MAX_STRING_LENGTH);
      
      // Generate a random string
      for (uint32_t j = 0; j < MAX_STRING_LENGTH-1; ++j) {
        str[j] = charset[char_dist(rng)];
      }
      // Ensure null termination
      str[MAX_STRING_LENGTH-1] = '\0';
    }
    
    CUDA_CHECK(hipMemcpyAsync(dbuf.d_keys, h_keys.data(), this_batch*sizeof(K), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(dbuf.d_values, h_vals.data(), this_batch*cfg.dim*sizeof(V), hipMemcpyHostToDevice, stream));
    table->insert_or_assign(this_batch, dbuf.d_keys, dbuf.d_values, nullptr, stream);  // Pass nullptr for scores
    inserted += this_batch;
    if (inserted % (1ull * 1024) == 0) {
      std::cout << "Preloaded " << inserted << " records..." << std::endl;
    }
  }
  CUDA_CHECK(hipStreamSynchronize(stream));

  // ------------------------------------------------------
  // Workload phase
  // ------------------------------------------------------
  std::cout << "Executing " << cfg.num_batch_ops << " batch operations..." << std::endl;
  double total_time = 0;
  Timer<double> timer;
  uint64_t total_ops = 0;

  // Create a vector to determine the type of each batch (true for read, false for write)
  std::vector<bool> batch_types(cfg.num_batch_ops);
  uint64_t num_read_batches = static_cast<uint64_t>(cfg.num_batch_ops * cfg.read_prop);
  std::fill(batch_types.begin(), batch_types.begin() + num_read_batches, true);
  std::fill(batch_types.begin() + num_read_batches, batch_types.end(), false);
  
  // Shuffle the batch types
  std::mt19937_64 rng_batch(cfg.seed);
  std::shuffle(batch_types.begin(), batch_types.end(), rng_batch);

  for (uint64_t batch = 0; batch < cfg.num_batch_ops; batch++) {
    bool is_read_batch = batch_types[batch];
    uint32_t batch_size = is_read_batch ? cfg.multiget_batch_size : cfg.multiset_batch_size;
    
    // Generate keys for this batch
    std::vector<K> keys; keys.reserve(batch_size);
    generate_keys_parallel(batch_size, *keygen, keys);

    std::cout << "Batch " << batch << ": " << (is_read_batch ? "READ" : "WRITE") 
              << " batch with " << keys.size() << " keys" << std::endl;

    if (is_read_batch) {
      // --- Read batch ---
      timer.start();
      CUDA_CHECK(hipMemcpyAsync(dbuf.d_keys, keys.data(), keys.size()*sizeof(K), hipMemcpyHostToDevice, stream));
      table->find(keys.size(), dbuf.d_keys, dbuf.d_values_out, dbuf.d_found, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      total_time += timer.getResult();
      total_ops += keys.size();
    } else {
      // --- Write batch ---
      // Generate random string values for write operations
      std::vector<CustomString> vals(batch_size * cfg.dim);
      for(uint32_t i=0; i<batch_size*cfg.dim; ++i) {
        CustomString& str = vals[i];
        memset(str.data, 0, MAX_STRING_LENGTH);
        for (uint32_t j = 0; j < MAX_STRING_LENGTH-1; ++j) {
          str[j] = charset[char_dist(rng)];
        }
        str[MAX_STRING_LENGTH-1] = '\0';
      }

      timer.start();
      CUDA_CHECK(hipMemcpyAsync(dbuf.d_keys, keys.data(), keys.size()*sizeof(K), hipMemcpyHostToDevice, stream));
      CUDA_CHECK(hipMemcpyAsync(dbuf.d_values, vals.data(), keys.size()*cfg.dim*sizeof(V), hipMemcpyHostToDevice, stream));
      table->insert_or_assign(keys.size(), dbuf.d_keys, dbuf.d_values, nullptr, stream);
      CUDA_CHECK(hipStreamSynchronize(stream));
      timer.end();
      total_time += timer.getResult();
      total_ops += keys.size();
    }

    std::cout << "Completed batch " << batch << ", total ops so far: " << total_ops << std::endl;
  }
  CUDA_CHECK(hipStreamSynchronize(stream));

  double secs = total_time;
  double ops_per_sec = total_ops / secs;
  double gbkv_per_sec = total_ops / secs / 1e9;

  std::cout << "total_ops,time_ms,ops_per_sec,GB-kv/s\n";
  std::cout << total_ops << "," << secs*1000.0 << "," << ops_per_sec << "," << gbkv_per_sec << std::endl;

  hipStreamDestroy(stream);
  return {secs, ops_per_sec, gbkv_per_sec};
}